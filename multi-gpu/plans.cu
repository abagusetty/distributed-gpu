#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <omp.h>
#include "cuda_help.h"
#include "plans.h"


void create_slab_pencil_plans(int NX, int NY, int NZ,
		       int LX, int LY, int LZ,
		       hipfftHandle *plan1D, hipfftHandle *plan2D){
  
  const int dimensions = 2;
  int dims[dimensions] = {NX, NY};    
  
  // Create plans for computing X-Y
  if (hipfftPlanMany(plan2D, dimensions, dims,
		    NULL, 1, 1, // in      
		    NULL, 1, 1, // out
		    HIPFFT_Z2Z, LZ) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Plan creation failed");
    exit(0);
  }
  
  int zdims[1] = {NZ};
  int inembed[1] = {NZ};
  int onembed[1] = {NZ};
  if ( hipfftPlanMany(plan1D, 1, zdims,
		     inembed, LX*LY, 1,   // in
		     onembed, 1, NZ,      // out
		     HIPFFT_Z2Z, LX*LY
		     ) != HIPFFT_SUCCESS) {
    fprintf(stderr, "CUFFT error: Plan creation failed");
    exit(0);
  }    
}


void destroy_slab_pencil_plans(hipfftHandle *plan1D, hipfftHandle *plan2D){
  hipfftDestroy(*plan1D);
  hipfftDestroy(*plan2D);
}
