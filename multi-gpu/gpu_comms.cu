#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <omp.h>
#include "cuda_help.h"
#include "gpu_comms.h"


void init_GPU_peer(int p, int tid, hipStream_t *streams){
  // enable peer to peer.
  for (int dd = 0; dd < p; dd++) {
    int access = 0;
    hipDeviceCanAccessPeer(&access, tid, dd);
    if (access){
        hipDeviceEnablePeerAccess(dd, 0);
        cudaCheckError();
    }
  }

  for (int s = 0; s < NUM_STREAMS; s++) {
    hipStreamCreateWithFlags(&streams[s], hipStreamNonBlocking);
    cudaCheckError();
  }

}

void finalize_GPU_peer(hipStream_t *streams){
  for (int s = 0; s < NUM_STREAMS; s++) {
    hipStreamDestroy(streams[s]);
    cudaCheckError();
  }  
}
